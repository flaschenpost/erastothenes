#include "hip/hip_runtime.h"
/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */

// Includes
#include <stdio.h>
#include <string.h>

#define ST int
#define SW sizeof(ST)*8

// Variables
ST * isComposite;
ST * d_isComposite;
int initPrimes[9] = {5,7,11,13,17,19,23,29,31};

void Cleanup(bool);

// Device code
__global__ void initPrim(ST * C, const int initPrimes[], int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for(int j = 0; j<24; j++){
        if(24*i+j < N){
            C[24*i+j] = 24*i+j - i;
        }
    }
}

// Host code
int main(int argc, char** argv)
{
    printf("Memory preset \n");
    int max = 600UL;

    size_t size = max; 
    
    size_t b_size = size;

    hipError_t error;

    // Allocate input vectors h_A and h_B in host memory
    isComposite = (ST*)malloc(size* sizeof(ST));
    if (isComposite == 0) Cleanup(false);

    int i;

    printf("Setting %i bytes\n", b_size);
    // Allocate vectors in device memory
    error = hipMalloc((void**)&d_isComposite, 8*b_size*sizeof(ST));
    if (error != hipSuccess) Cleanup(false);
    
    printf("init 0.04 at %lu  %i bytes to 0\n", d_isComposite, b_size*sizeof(ST));
    hipMemset(d_isComposite, (int)1, 8*b_size*sizeof(ST));
    if (error != hipSuccess) Cleanup(false);

    // Invoke kernel
   
    int threadsPerBlock = 16;
    int blocksPerGrid = (b_size*8 + threadsPerBlock - 1) / threadsPerBlock/24;
    initPrim<<<blocksPerGrid, threadsPerBlock>>>(d_isComposite, initPrimes, b_size*8);
    error = hipGetLastError();
    printf("error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock, b_size);
    if (error != hipSuccess) Cleanup(false);
#ifdef _DEBUG
    error = hipDeviceSynchronize();
    if (error != hipSuccess) Cleanup(false);
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    error = hipMemcpy(isComposite, d_isComposite, b_size*sizeof(int), hipMemcpyDeviceToHost);
    printf("2. error = %d / %d\n", error, hipSuccess);
    if (error != hipSuccess) Cleanup(false);
    
    for (i = 0; i < b_size; ++i) {
        printf(" . %lu : ", isComposite[i]);
        /*
        if(isComposite[i/SW] & (1UL << (i%SW))){
            printf("_1\n");
        }
        else{
            printf("_0\n");
        }
        */
    }
    Cleanup(true);
}

void Cleanup(bool noError)
{
    hipError_t error;
        
    // Free device memory
    if (d_isComposite)
        error = hipFree(d_isComposite);
    // Free host memory
    if (isComposite)
        free(isComposite);
        
    error = hipDeviceReset();
    
    if (error != hipSuccess)
        printf("Function call failed\nTest FAILED\n");
    
        printf("\nPress ENTER to exit...\n");
        fflush( stdout);
        fflush( stderr);
        getchar();

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

