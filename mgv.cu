#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <string.h>

#define ST unsigned int
#define SW (sizeof(ST)*8)

#define PRT unsigned short

// per thread, so only small part of the card-memory, in Bytes
#define CUDABLOCK 512
// #define FIRSTPRIMES 9
#define FIRSTPRIMES 16

// Variables
ST * isComposite;
__shared__ ST * d_isComposite;

void Cleanup(bool);
PRT initPrimes[FIRSTPRIMES] = {5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61};

// Device code
__global__ void initPrim(ST * C, const int offset, unsigned long N)
{
    unsigned long i = blockDim.x * blockIdx.x + threadIdx.x;
    __const__ PRT d_initPrimes[FIRSTPRIMES] = {5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61};
    // bits
    unsigned long block = CUDABLOCK * SW;

    unsigned long base = block*i + offset ;
    unsigned long max = base + block;

    for(int l = 0; l < FIRSTPRIMES; l++){
        PRT p = d_initPrimes[l];
        // unsigned long back = base % (2*p);
        unsigned long j,k;
        j = (p*p-5)/2 - (p*p-5)/6 ;
        if(p%3 == 1){
            k = ((p+4)*p-5)/2 - (((p+4)*p-5)/2)/3;
        }
        else{
            k = ((p+2)*p-5)/2 - (((p+2)*p-5)/2)/3;
        }

        if(base > 0){
            j = base + j % (2*p) - base % (2*p);
            k = base + k % (2*p) - base % (2*p);
            if(j < base){
                j += 2*p;
            }
            if(k < base){
                k += 2*p;
            }
        }
        if(k < j){
            unsigned long tmp = j;
            j = k;
            k = tmp;
        }

        /*
        if(p == 11){
            C[5*i+0] = base;
            C[5*i+1] = p;
            C[5*i+2] = max;
            C[5*i+3] = j;
            C[5*i+4] = k;
            return;
        }
         // */
        if(j < base){
            j += 2*p;
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            k += 2*p;
        }

        if(max > N){
            max = N;
        }

        while(j <= max - 2*p){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            j += 2*p;
            k += 2*p;
        }
        if(j < max){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
        }
        if(k < max){
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
        }
    }
}

// Host code
int main(int argc, char** argv)
{
    int threadsPerBlock = 64;
    int blocksPerGrid=1;

    printf("Memory preset \n");
    size_t size_bytes = blocksPerGrid * threadsPerBlock* CUDABLOCK; 

    // Bits
    unsigned long max = size_bytes*8;

    hipError_t error;

    // Allocate input vectors h_A and h_B in host memory
    isComposite = (ST*)malloc(size_bytes);
    if (isComposite == 0) Cleanup(false);

    // initPrimF(0, 0, 0, isComposite, 0, max);
    // initPrimF(0, 0, 1, isComposite, 0, max);

    unsigned long i;

    printf("Setting %i bytes\n", size_bytes);
    // Allocate vectors in device memory
    error = hipMalloc((void**)&d_isComposite, size_bytes);
    if (error != hipSuccess){
        printf("mal isC: error = %d / %d : \n", error, hipSuccess );
        Cleanup(false);
    }

    printf("init at %lu  %lu bytes to 0\n", d_isComposite, size_bytes);
    hipMemset(d_isComposite, 0, size_bytes);
    if (error != hipSuccess) Cleanup(false);

    // Invoke kernel

    initPrim<<<blocksPerGrid, threadsPerBlock>>>(d_isComposite, 0, max);
    error = hipGetLastError();
    if (error != hipSuccess){
        printf("error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
        Cleanup(false);
    }
// #ifdef _DEBUG
    error = hipDeviceSynchronize();
    printf("sync error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
    if (error != hipSuccess) Cleanup(false);
// #endif

    // Copy result from device memory to host memory
    error = hipMemcpy(isComposite, d_isComposite, size_bytes, hipMemcpyDeviceToHost);
    if (error != hipSuccess){
        printf("2. error = %d / %d\n", error, hipSuccess);
        Cleanup(false);
    }
    // */
    // check(max, isComposite);

    Cleanup(true);
}
void check(long max, ST *isComposite){
    for (long i = 0; i < max; ++i) {
        unsigned long long p = i;
        p += i/2; p<<=1; p += 5;
        int p_is_comp = 0, p_is_marked = 0;

        /*
        printf("hu %d %lu\n", i, isComposite[i]);
        if(i > 90){
            break;
        }
        continue;
        if(isComposite[i/SW] & (1UL << (i%SW))){
            printf("%4d %6d _1\n", i, 5+2*(i+i/2));
        }
        else{
            printf("%4d %6d _0\n", i, 5+2*(i+i/2));
        }
        // */
        for ( int j = 0; j < FIRSTPRIMES; j++){
            if (p % initPrimes[j] == 0 && p > initPrimes[j]){
                p_is_comp = initPrimes[j];
                break;
            }
        }
        p_is_marked = isComposite[i/SW] & (1UL << (i%SW));
        if( p_is_comp && ! p_is_marked){
            printf(" %lu: %llu comp %llu but unmarked / %lu\n", i, p, p_is_comp, max);
            exit(1);
        }
        if( !p_is_comp && p_is_marked){
            printf(" %lu: %llu not comp %llu but marked / %lu\n", i, p, p_is_comp, max);
            exit(1);
        }
    }
}

void Cleanup(bool noError)
{
    hipError_t error;

    // Free device memory
    if (d_isComposite)
        error = hipFree(d_isComposite);
    // Free host memory
    if (isComposite)
        free(isComposite);

    error = hipDeviceReset();

    if (error != hipSuccess)
        printf("Function call failed\nTest FAILED\n");

    exit(0);
}
