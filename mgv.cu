#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <string.h>

#define ST uint8
#define SW (sizeof(ST)*8)

#define PRT unsigned short

// per thread, so only small part of the card-memory, in Bytes
#define CUDABLOCK 2048
// #define FIRSTPRIMES 9
#define FIRSTPRIMES 16

// Variables
ST * isComposite;
__device__ ST * d_isComposite;

void Cleanup(bool);
PRT initPrimes[FIRSTPRIMES] = {5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61};

// Device code
__global__ void initPrim(ST * C, const int offset, unsigned long N)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    __const__ PRT d_initPrimes[FIRSTPRIMES] = {5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61};
    // bits
    unsigned long block = CUDABLOCK * SW;

    unsigned long base = block*i + offset ;
    unsigned long max = block*(i+1);

    for(int l = 0; l < FIRSTPRIMES; l++){
        unsigned long p = d_initPrimes[l];
        unsigned long j,k;
        j = (p*p-5)/2 - (p*p-5)/6 ;
        if(p%3 == 1){
            k = ((p+4)*p-5)/2 - (((p+4)*p-5)/2)/3;
        }
        else{
            k = ((p+2)*p-5)/2 - (((p+2)*p-5)/2)/3;
        }

        if(base > 0){
            j = base + j % (2*p) - base % (2*p);
            k = base + k % (2*p) - base % (2*p);
            if(j < base){
                j += 2*p;
            }
            if(k < base){
                k += 2*p;
            }
        }
        if(k < j){
            unsigned long tmp = j;
            j = k;
            k = tmp;
        }

        if(max > N){
            max = N;
        }

        j -= offset;
        k -= offset;

        while(j <= max - 2*p){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            j += 2*p;
            k += 2*p;
        }
        if(j < max){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
        }
        if(k < max){
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
        }
    }
}

// Host code
int main(int argc, char** argv)
{
    int threadsPerBlock = 128;
    int blocksPerGrid=1;

    int loops = 4;

    size_t size_ST = blocksPerGrid * threadsPerBlock* CUDABLOCK;
    size_t size_bytes = size_ST * sizeof(ST); 

    // Bits
    unsigned long max = size_bytes*8;

    hipError_t error;

    // Allocate input vectors h_A and h_B in host memory
    isComposite = (ST*)malloc(loops*size_bytes*sizeof(ST));
    if (isComposite == 0) Cleanup(false);
    printf("Memory preset %lu bits\n", loops*size_bytes*sizeof(ST)*8);

    unsigned long i;

    // printf("Setting %i bytes\n", size_bytes);
    // Allocate vectors in device memory
    error = hipMalloc((void**)&d_isComposite, size_bytes*sizeof(ST));
    if (error != hipSuccess){
        printf("mal isC: error = %d / %d : \n", error, hipSuccess );
        Cleanup(false);
    }

    for(int lp = 0; lp < loops; lp++){
        printf("lp %d: init at %lu  %lu %l bytes to 0\n", lp, d_isComposite, max, isComposite);
        hipMemset(d_isComposite, 0, size_bytes);
        if (error != hipSuccess) Cleanup(false);

        // Invoke kernel

        initPrim<<<blocksPerGrid, threadsPerBlock>>>(d_isComposite, lp*max, max);
        error = hipGetLastError();
        if (error != hipSuccess){
            printf("error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
            Cleanup(false);
        }
        error = hipDeviceSynchronize();
        if (error != hipSuccess){
            printf("sync error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
            Cleanup(false);
        }
        error = hipMemcpy(isComposite+lp*size_ST, d_isComposite, size_bytes, hipMemcpyDeviceToHost);
        if (error != hipSuccess){
            printf("2. error = %d / %d\n", error, hipSuccess);
            Cleanup(false);
        }
        // printf("%lu %lu %lu %lu \n", isComposite[lp*size_bytes], isComposite[lp*size_bytes+1], isComposite[lp*size_bytes+2], isComposite[lp*size_bytes+3]);
    }

    //*
    for (i = 0; i < 0*loops*max; ++i) {
        unsigned long long p = i;
        p += i/2; p<<=1; p += 5;
        int p_is_comp = 0, p_is_marked = 0;

        /*
        if(isComposite[i/SW] & (1UL << (i%SW))){
            printf("%4d %6d _1\n", i, 5+2*(i+i/2));
        }
        else{
            printf("%4d %6d _0\n", i, 5+2*(i+i/2));
        }
        // */
        for ( int j = 0; j < FIRSTPRIMES; j++){
            if (p % initPrimes[j] == 0 && p > initPrimes[j]){
                p_is_comp = initPrimes[j];
                break;
            }
        }
        p_is_marked = isComposite[i/SW] & (1UL << (i%SW));
        if( p_is_comp && ! p_is_marked){
            printf(" %lu: %llu comp %llu but unmarked \n", i, p, p_is_comp);
            Cleanup(1);
        }
        if( !p_is_comp && p_is_marked){
            printf(" %lu: %llu not comp %llu but marked \n", i, p, p_is_comp);
            Cleanup(1);
        }
    }
    Cleanup(true);
}

void Cleanup(bool noError)
{
    hipError_t error;
    printf("cleanup!\n");

    // Free device memory
    if (d_isComposite)
        error = hipFree(d_isComposite);
    // Free host memory
    if (isComposite)
        free(isComposite);

    error = hipDeviceReset();

    if (error != hipSuccess)
        printf("Function call failed\nTest FAILED\n");

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

