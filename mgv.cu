#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <string.h>

#define ST unsigned int
#define SW (sizeof(ST)*8)

#define PRT unsigned long

// per thread, so only small part of the card-memory, in Bytes
#define CUDABLOCK 512
// #define FIRSTPRIMES 9
#define FIRSTPRIMES 16

// Variables
ST * isComposite;
ST * d_isComposite;

void Cleanup(bool);
PRT initPrimes[FIRSTPRIMES] = {5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61};

// Device code
__global__ void initPrim(ST * C, const int offset, unsigned long N)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    __const__ PRT d_initPrimes[FIRSTPRIMES] = {5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61};
    // bits
    unsigned long block = CUDABLOCK * SW;

    unsigned long base = block*i + offset ;
    unsigned long max = base + block;

    for(int l = 0; l < FIRSTPRIMES; l++){
        unsigned long p = d_initPrimes[l];
        unsigned long back = base % (2*p);
        unsigned long j,k;
        j = (p*p-5)/2 - (p*p-5)/6 ;
        if(p%3 == 1){
            k = ((p+4)*p-5)/2 - (((p+4)*p-5)/2)/3;
        }
        else{
            k = ((p+2)*p-5)/2 - (((p+2)*p-5)/2)/3;
        }
        unsigned long diff = k-j;

        if(i + offset > 0){
            j = base + (p-5)/2 - (p-5)/6;
            k = diff + j;
        }

        /*
        C[10*i+0] = base;
        C[10*i+1] = i;
        C[10*i+2] = j;
        C[10*i+3] = k;
        C[10*i+4] = p;
        return;
         */
        if(j < base){
            j += 2*p;
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            k += 2*p;
        }

        if(max > N){
            max = N;
        }

        while(j <= max - 2*p){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            j += 2*p;
            k += 2*p;
        }
        if(j < max){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
        }
        if(k < max){
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
        }
    }
}

// Host code
int main(int argc, char** argv)
{
    int threadsPerBlock = 32;
    int blocksPerGrid=1;

    printf("Memory preset \n");
    size_t size_bytes = blocksPerGrid * threadsPerBlock* CUDABLOCK; 

    // Bits
    unsigned long max = size_bytes*8;

    hipError_t error;

    // Allocate input vectors h_A and h_B in host memory
    isComposite = (ST*)malloc(size_bytes);
    if (isComposite == 0) Cleanup(false);

    unsigned long i;

    printf("Setting %i bytes\n", size_bytes);
    // Allocate vectors in device memory
    error = hipMalloc((void**)&d_isComposite, size_bytes);
    if (error != hipSuccess){
        printf("mal isC: error = %d / %d : \n", error, hipSuccess );
        Cleanup(false);
    }

    printf("init 0.04 at %lu  %lu bytes to 0\n", d_isComposite, max);
    hipMemset(d_isComposite, 0, size_bytes);
    if (error != hipSuccess) Cleanup(false);

    // Invoke kernel

    initPrim<<<blocksPerGrid, threadsPerBlock>>>(d_isComposite, 0, max);
    error = hipGetLastError();
    if (error != hipSuccess){
        printf("error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
        Cleanup(false);
    }
// #ifdef _DEBUG
    error = hipDeviceSynchronize();
    printf("sync error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
    if (error != hipSuccess) Cleanup(false);
// #endif

    // Copy result from device memory to host memory
    error = hipMemcpy(isComposite, d_isComposite, size_bytes, hipMemcpyDeviceToHost);
    if (error != hipSuccess){
        printf("2. error = %d / %d\n", error, hipSuccess);
        Cleanup(false);
    }
    // */

    //*
    for (i = 0; i < max; ++i) {
        unsigned long long p = i;
        p += i/2; p<<=1; p += 5;
        int p_is_comp = 0, p_is_marked = 0;

        /*
        printf("hu %d %lu\n", i, isComposite[i]);
        if(i > 90){
            break;
        }
        continue;
        if(isComposite[i/SW] & (1UL << (i%SW))){
            printf("%4d %6d _1\n", i, 5+2*(i+i/2));
        }
        else{
            printf("%4d %6d _0\n", i, 5+2*(i+i/2));
        }
        // */
        for ( int j = 0; j < FIRSTPRIMES; j++){
            if (p % initPrimes[j] == 0 && p > initPrimes[j]){
                p_is_comp = initPrimes[j];
                break;
            }
        }
        p_is_marked = isComposite[i/SW] & (1UL << (i%SW));
        if( p_is_comp && ! p_is_marked){
            printf(" %lu: %llu comp %llu but unmarked \n", i, p, p_is_comp);
            exit(1);
        }
        if( !p_is_comp && p_is_marked){
            printf(" %lu: %llu not comp %llu but marked \n", i, p, p_is_comp);
            exit(1);
        }
    }
    Cleanup(true);
}

void Cleanup(bool noError)
{
    hipError_t error;

    // Free device memory
    if (d_isComposite)
        error = hipFree(d_isComposite);
    // Free host memory
    if (isComposite)
        free(isComposite);

    error = hipDeviceReset();

    if (error != hipSuccess)
        printf("Function call failed\nTest FAILED\n");

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

