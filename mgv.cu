#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <string.h>

#define ST unsigned int
#define SW sizeof(ST)*8

// per thread, so only small part of the card-memory, in Bytes
#define CUDABLOCK 512
// #define FIRSTPRIMES 9
#define FIRSTPRIMES 11

// Variables
ST * isComposite;
unsigned int * d_isComposite;
ST * d_initPrimes;

unsigned int initPrimes[FIRSTPRIMES] = {5,7,11,13,17,19,23,29,31,37,41};

void Cleanup(bool);

// check
void initPrim(int bDim, int bIdx, int thIdx, ST * C, const int offset, const unsigned int *d_initPrimes, int N)
{
    unsigned int i = bDim * bIdx + thIdx;

    unsigned long block = CUDABLOCK * SW;

    unsigned long base = block*i + offset ;
    unsigned long max = base + block;

    for(int l = 0; l < FIRSTPRIMES; l++){
        unsigned long p = d_initPrimes[l];
        unsigned long back = base % (2*p);
        unsigned int j,k;
        j = (p*p-5)/2 - (p*p-5)/6 ;
        if(p%3 == 1){
            k = ((p+4)*p-5)/2 - (((p+4)*p-5)/2)/3;
        }
        else{
            k = ((p+2)*p-5)/2 - (((p+2)*p-5)/2)/3;
        }
        if(i + offset > 0){
            j += base - back;
            k += base - back;
        }


        if(j < base){
            j += 2*p;
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            k += 2*p;
        }

        if(max > N){
            max = N;
        }

        printf("%d, %lu, %lu   %lu %lu \n", i, j, k, block, back);
        while(j <= max - 2*p){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            j += 2*p;
            k += 2*p;
        }
        if(j < max){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
        }
        if(k < max){
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
        }
    }
}

// Device code
__global__ void initPrim(ST * C, const int offset, const unsigned int *d_initPrimes, int N)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned long block = CUDABLOCK * SW;

    unsigned long base = block*i + offset ;
    unsigned long max = base + block;

    for(int l = 0; l < FIRSTPRIMES; l++){
        unsigned long p = d_initPrimes[l];
        unsigned long back = base % (2*p);
        unsigned int j,k;
        j = (p*p-5)/2 - (p*p-5)/6 ;
        if(p%3 == 1){
            k = ((p+4)*p-5)/2 - (((p+4)*p-5)/2)/3;
        }
        else{
            k = ((p+2)*p-5)/2 - (((p+2)*p-5)/2)/3;
        }
        unsigned long diff = k-j;

        if(i + offset > 0){
            j = base + (p-5)/2 - (p-5)/6;
            k = diff + j;
        }

        /*
        C[10*i+0] = base;
        C[10*i+1] = i;
        C[10*i+2] = j;
        C[10*i+3] = k;
        C[10*i+4] = p;
        return;
         */
        if(j < base){
            j += 2*p;
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            k += 2*p;
        }

        if(max > N){
            max = N;
        }

        while(j <= max - 2*p){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
            j += 2*p;
            k += 2*p;
        }
        if(j < max){
            C[j / SW] |= ((ST)1 << ((j) % SW)); 
        }
        if(k < max){
            C[k / SW] |= ((ST)1 << ((k) % SW)); 
        }
    }
}

// Host code
int main(int argc, char** argv)
{
    printf("Memory preset \n");
    int max = 32*8* CUDABLOCK;

    size_t size = max; 

    size_t b_size = size;

    hipError_t error;

    // Allocate input vectors h_A and h_B in host memory
    isComposite = (ST*)malloc(size* sizeof(ST));
    if (isComposite == 0) Cleanup(false);

    unsigned long i;

    printf("Setting %i bytes\n", b_size);
    // Allocate vectors in device memory
    // error = hipMalloc((void**)&d_isComposite, SW*(b_size)*sizeof(ST));
    error = hipMalloc((void**)&d_isComposite, max*sizeof(ST));
    if (error != hipSuccess){
        printf("mal isC: error = %d / %d : \n", error, hipSuccess );
        Cleanup(false);
    }

    error = hipMalloc((void**)&d_initPrimes, FIRSTPRIMES*sizeof(ST));
    if (error != hipSuccess){
        printf("mal ini: error = %d / %d : \n", error, hipSuccess );
        Cleanup(false);
    }


    error = hipMemcpy(d_initPrimes, initPrimes, FIRSTPRIMES*sizeof(unsigned int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("2. error = %d / %d\n", error, hipSuccess);
        Cleanup(false);
    }

    printf("init 0.04 at %lu  %lu bytes to 0\n", d_isComposite, max);
    hipMemset(d_isComposite, 0, max*sizeof(ST));
    if (error != hipSuccess) Cleanup(false);

    // Invoke kernel

    int threadsPerBlock = 16;
    int blocksPerGrid=1;

    initPrim<<<blocksPerGrid, threadsPerBlock>>>(d_isComposite, 0, d_initPrimes, max);
    error = hipGetLastError();
    if (error != hipSuccess){
        printf("error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
        Cleanup(false);
    }
// #ifdef _DEBUG
    error = hipDeviceSynchronize();
    printf("sync error = %d / %d : %dx%d / %d\n", error, hipSuccess, blocksPerGrid, threadsPerBlock);
    if (error != hipSuccess) Cleanup(false);
// #endif

    // Copy result from device memory to host memory
    error = hipMemcpy(isComposite, d_isComposite, max, hipMemcpyDeviceToHost);
    if (error != hipSuccess){
        printf("2. error = %d / %d\n", error, hipSuccess);
        Cleanup(false);
    }
    // */

    //*
    for (i = 0; i < max; ++i) {
        unsigned long long p = i;
        p += i/2; p<<=1; p += 5;
        int p_is_comp = 0, p_is_marked = 0;

        /*
        printf("hu %d %lu\n", i, isComposite[i]);
        if(i > 90){
            break;
        }
        continue;
        /*
        if(isComposite[i/SW] & (1UL << (i%SW))){
            printf("%4d %6d _1\n", i, 5+2*(i+i/2));
        }
        else{
            printf("%4d %6d _0\n", i, 5+2*(i+i/2));
        }
        // */
        for ( int j = 0; j < FIRSTPRIMES; j++){
            if (p % initPrimes[j] == 0 && p > initPrimes[j]){
                p_is_comp = initPrimes[j];
                break;
            }
        }
        p_is_marked = isComposite[i/SW] & (1UL << (i%SW));
        if( p_is_comp && ! p_is_marked){
            printf(" %lu: %llu comp %llu but unmarked \n", i, p, p_is_comp);
            exit(1);
        }
        if( !p_is_comp && p_is_marked){
            printf(" %lu: %llu not comp %llu but marked \n", i, p, p_is_comp);
            exit(1);
        }
    }
    Cleanup(true);
}

void Cleanup(bool noError)
{
    hipError_t error;

    // Free device memory
    if (d_isComposite)
        error = hipFree(d_isComposite);
    // Free host memory
    if (isComposite)
        free(isComposite);

    error = hipDeviceReset();

    if (error != hipSuccess)
        printf("Function call failed\nTest FAILED\n");

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

